#include "hip/hip_runtime.h"
#include "roof.h"

#define BLOCKSIZE 1024

__global__ void saxpy(int n, float a, float *x, float *y)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < n)
        y[i] = a * x[i] + y[i];
}


extern "C"
void gpu_axpy(int n, float a, float b, float * x_in, float * y_in,
              struct roof_args *args)
{
    float *x, *y;
    size_t nbytes;

    int r, r_max;

    hipEvent_t start, stop;
    float msec, sec;

    // Timer setup
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nbytes = n * sizeof(float);
    hipMalloc(&x, nbytes);
    hipMalloc(&y, nbytes);

    hipMemcpy(x, x_in, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(y, y_in, nbytes, hipMemcpyHostToDevice);

    r_max = 1;
    hipEventRecord(start);
    for (r = 0; r < r_max; r++) {
        saxpy<<<1 + n / BLOCKSIZE, BLOCKSIZE>>>(n, a, x, y);
    }
    hipEventRecord(stop);

    hipMemcpy(y_in, y, nbytes, hipMemcpyDeviceToHost);

    hipFree(x);
    hipFree(y);

    hipEventElapsedTime(&msec, start, stop);
    sec = msec / 1000.f;

    args->runtime = sec;
    args->flops = 2. * r_max * n / sec;
    args->bw_load = 2. * r_max * nbytes / sec;
    args->bw_store = 1. * r_max * nbytes / sec;
}
