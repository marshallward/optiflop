#include "hip/hip_runtime.h"
#include "roof.h"
#include <stdio.h>

#define MAXCORES 2
#define MAXTHREADS 800


__global__ void saxpy(int n, double a, double *x, double *y)
{
    int i0 = MAXCORES * (blockDim.x * blockIdx.x + threadIdx.x);
    for (int i = i0; i < min(i0 + MAXCORES, n); i++)
        y[i] = a * x[i] + y[i];
}


extern "C"
void gpu_axpy(int n, double a, double b, double * x_in, double * y_in,
              struct roof_args *args)
{
    double *x, *y;
    size_t nbytes;

    int r, r_max;
    int nthreads, nblocks;

    hipEvent_t start, stop;
    float msec, sec;

    volatile double sum;

    // Timer setup
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nbytes = n * sizeof(double);
    hipMalloc(&x, nbytes);
    hipMalloc(&y, nbytes);

    hipMemcpy(x, x_in, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(y, y_in, nbytes, hipMemcpyHostToDevice);

    nthreads = min(1 + (n - 1) / MAXCORES, MAXTHREADS);
    nblocks = 1 + (n - 1) / (MAXTHREADS * MAXCORES);

    //printf("  ncores: %i\n", MAXCORES);
    //printf("nthreads: %i\n", nthreads);
    //printf(" nblocks: %i\n", nblocks);

    r_max = 1;
    hipEventRecord(start);
    for (r = 0; r < r_max; r++) {
        saxpy<<<nblocks,nthreads>>>(n, a, x, y);
    }
    hipEventRecord(stop);

    hipMemcpy(y_in, y, nbytes, hipMemcpyDeviceToHost);

    /* Not yet confident this is working, so check the sum. */
    /* Also ensures that the value is touched and won't be optimized out. */
    /* TODO: Later, we can rely on `volatile` and drop this sum. */
    sum = 0.;
    for (int i = 0; i < n; i++) sum += y_in[i];
    if (sum != 4. * n) {
        printf("ERROR: Sum (%f\n does not match!\n", sum);
        exit(1);
    }

    hipFree(x);
    hipFree(y);

    hipEventElapsedTime(&msec, start, stop);
    sec = msec / 1000.f;

    args->runtime = sec;
    args->flops = 2. * r_max * n / sec;
    args->bw_load = 2. * r_max * nbytes / sec;
    args->bw_store = 1. * r_max * nbytes / sec;
}
