#include "hip/hip_runtime.h"
#include "roof.h"

/* TODO: V100-specific numbers; how to generalize? */
#define NCORES 32
#define NBLOCKS 160
#define NTHREADS 128

__global__ void kadd(long r_max, float *sum)
{
    const float eps = 1e-6f;
    float reg[NCORES];
    long r;
    int i;

    for (i = 0; i < NCORES; i++)
        reg[i] = 1.f;

    for (r = 0; r < r_max; r++)
        for (i = 0; i < NCORES; i++)
            reg[i] = reg[i] + eps;

    *sum = 0.f;
    for (i = 0; i < NCORES; i++) *sum = *sum + reg[i];
}


__global__ void kmul(long r_max, float *sum)
{
    const float alpha = 1.f + 1e-6f;
    float reg[NCORES];
    long r;
    int i;

    for (i = 0; i < NCORES; i++)
        reg[i] = 1.f;

    for (r = 0; r < r_max; r++)
        for (i = 0; i < NCORES; i++)
            reg[i] = reg[i] * alpha;

    *sum = 0.f;
    for (i = 0; i < NCORES; i++) *sum = *sum + reg[i];
}


__global__ void kfma(long r_max, float *sum)
{
    const float eps = 1e-6f;
    const float alpha = 1.f + 1e-6f;
    float reg[NCORES];
    long r;
    int i;

    for (i = 0; i < NCORES; i++)
        reg[i] = 1.f;

    for (r = 0; r < r_max; r++)
        for (i = 0; i < NCORES; i++)
            reg[i] = alpha * reg[i] + eps;

    *sum = 0.f;
    for (i = 0; i < NCORES; i++) *sum = *sum + reg[i];
}


extern "C"
void gpu_add(void *args_in)
{
    struct roof_args *args;     // args
    hipEvent_t start, stop;
    long r_max;
    float sum, *gpu_sum;
    float msec, runtime;

    args = (struct roof_args *) args_in;

    r_max = 1;
    hipMalloc(&gpu_sum, sizeof(float));

    /* TODO: Move timer to kernel and use clock64() */
    hipEventCreate(&start);
    hipEventCreate(&stop);

    *(args->runtime_flag) = 0;
    do {
        hipEventRecord(start);
        kadd<<<NBLOCKS,NTHREADS>>>(r_max, gpu_sum);
        hipEventRecord(stop);

        // Get results
        hipMemcpy(&sum, gpu_sum, sizeof(float), hipMemcpyDeviceToHost);

        hipEventElapsedTime(&msec, start, stop);
        runtime = msec / 1000.f;

        if (runtime > args->min_runtime)
            *(args->runtime_flag) = 1;
        // TODO: Set mutex before write?

        // TODO: barrier?

        if (! *(args->runtime_flag)) r_max *= 2;
    } while (! *(args->runtime_flag));

    args->runtime = runtime;
    args->flops = (float) NBLOCKS * NTHREADS * NCORES * r_max / runtime;
    //args->flops = (float) 2 * NBLOCKS * NTHREADS * NCORES * r_max / runtime;
    args->bw_load = 0;
    args->bw_store = 0;

    hipFree(gpu_sum);
}
