#include "hip/hip_runtime.h"
#include "roof.h"
#include <stdio.h>

#define MAXCORES 1
#define MAXTHREADS 64

__global__ void delsq(int n, double a, double b, double *x, double *y)
{
    int i0 = MAXCORES * (blockDim.x * blockIdx.x + threadIdx.x);
    for (int i = i0; i < min(i0 + MAXCORES, n); i++)
        //if (i > 0 && i < n-1)
        //    y[i] = a * x[i] + b * (x[i-1] + x[i+1]);
        //if (i > 8 && i < n-8)
        //    y[i] = a * x[i] + b * (x[i-8] + x[i+8]);
        if (i > 0 && i < n-1)
            y[i] = a * x[i] + a * x[i+1];
}


__global__ void copy(int n, double a, double b, double *x, double *y)
{
    int i0 = MAXCORES * (blockDim.x * blockIdx.x + threadIdx.x);
    for (int i = i0; i < min(i0 + MAXCORES, n); i++)
        //x[i] = y[i];
        //y[i] = x[i] + y[i];
        y[i] = a * x[i];
}


extern "C"
void gpu_delsq(int n, double a, double b, double * x_in, double * y_in,
              struct roof_args *args)
{
    double *x, *y;
    size_t nbytes;

    long r_max;
    int nthreads, nblocks;

    hipEvent_t start, stop;
    float msec, sec;

    volatile double sum;

    // Timer setup
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nbytes = n * sizeof(double);
    hipMalloc(&x, nbytes);
    hipMalloc(&y, nbytes);

    hipMemcpy(x, x_in, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(y, y_in, nbytes, hipMemcpyHostToDevice);

    nthreads = min(1 + (n - 1) / MAXCORES, MAXTHREADS);
    nblocks = 1 + (n - 1) / (MAXTHREADS * MAXCORES);

    //printf("  ncores: %i\n", MAXCORES);
    //printf("nthreads: %i\n", nthreads);
    //printf(" nblocks: %i\n", nblocks);

    r_max = 1;
    *(args->runtime_flag) = 0;
    do {
        hipEventRecord(start);
        for (long r = 0; r < r_max; r++) {
            delsq<<<nblocks,nthreads>>>(n, a, b, x, y);
            //copy<<<nblocks,nthreads>>>(n, a, b, x, y);
        }
        hipEventRecord(stop);
        hipMemcpy(y_in, y, nbytes, hipMemcpyDeviceToHost);

        hipEventElapsedTime(&msec, start, stop);
        sec = msec / 1000.f;

        if (sec > args->min_runtime)
            *(args->runtime_flag) = 1;
        else
            r_max *= 2;

    } while (!*(args->runtime_flag));

    //sum = 0.;
    //for (int i = 0; i < n; i++) sum += y_in[i];
    //if (sum != n) {
    //    printf("ERROR: Sum %f\n does not match!\n", sum);
    //    exit(1);
    //}

    hipFree(x);
    hipFree(y);

    hipEventElapsedTime(&msec, start, stop);
    sec = msec / 1000.f;

    args->runtime = sec;
    //args->flops = 4. * r_max * n / sec;
    //args->bw_load = 1. * r_max * nbytes / sec;
    //args->bw_store = 1. * r_max * nbytes / sec;
    args->flops = 2. * r_max * n / sec;
    args->bw_load = 1. * r_max * nbytes / sec;
    args->bw_store = 1. * r_max * nbytes / sec;
}
