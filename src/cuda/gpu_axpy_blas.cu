#include "roof.h"
#include <stdio.h>
#include "stopwatch.h"

#include <hipblas.h>

#define MAXCORES 1
#define MAXTHREADS 64

extern "C"
void gpu_axpy_blas(int n, double a, double b, double * x_in, double * y_in,
                   struct roof_args *args)
{
    double *x, *y;
    size_t nbytes;

    long r_max;

    hipEvent_t start, stop;
    hipError_t error;
    float msec, sec;

    volatile double sum;

    // Timer setup
    hipEventCreate(&start);
    hipEventCreate(&stop);

    nbytes = n * sizeof(double);
    hipMalloc(&x, nbytes);
    hipMalloc(&y, nbytes);

    hipMemcpy(x, x_in, nbytes, hipMemcpyHostToDevice);
    hipMemcpy(y, y_in, nbytes, hipMemcpyHostToDevice);

    /* Apparently cuBLAS setup time is horrendous, so we need to run it
       at least once before starting the time... */
    hipblasDaxpy(n, a, x, 1, y, 1);

    r_max = 1;
    *(args->runtime_flag) = 0;
    do {
        hipEventRecord(start);
        for (long r = 0; r < r_max; r++) {
            hipblasDaxpy(n, a, x, 1, y, 1);
        }
        hipEventRecord(stop);
        hipEventSynchronize(stop);
        hipMemcpy(y_in, y, nbytes, hipMemcpyDeviceToHost);

        hipEventElapsedTime(&msec, start, stop);
        sec = msec / 1000.f;

        if (sec > args->min_runtime)
            *(args->runtime_flag) = 1;
        else
            r_max *= 2;

    } while (!*(args->runtime_flag));

    //sum = 0.;
    //for (int i = 0; i < n; i++) sum += y_in[i];
    //if (sum != n) {
    //    printf("ERROR: Sum %f\n does not match!\n", sum);
    //    exit(1);
    //}

    hipFree(x);
    hipFree(y);

    args->runtime = sec;
    args->flops = 2. * r_max * n / sec;
    args->bw_load = 2. * r_max * nbytes / sec;
    args->bw_store = 1. * r_max * nbytes / sec;
}
