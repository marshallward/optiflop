#include "roof.h"

// testing...
#include <unistd.h> // sleep

extern "C"
void gpu_avx(void *args_in)
{
    /* Thread input */
    struct roof_args *args;

    hipEvent_t start, stop;
    float msec, sec;

    args = (struct roof_args *) args_in;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start);
    /* TODO: Actual work! */
    sleep(1);
    hipEventRecord(stop);

    hipEventElapsedTime(&msec, start, stop);
    sec = msec / 1000.f;

    /* TODO: Actual work! */
    args->runtime = sec;
    args->flops = 0.;
    args->bw_load = 0;
    args->bw_store =0;
}
